#include "hip/hip_runtime.h"
/*
 *  Copyright 2011-2014 Maxim Milakov
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include "max_subsampling_layer_testing_schema.h"

#include "../neural_network_exception.h"
#include "../max_subsampling_layer.h"
#include "max_subsampling_layer_tester_cuda.cuh"

#include <boost/format.hpp>

namespace nnforge
{
	namespace cuda
	{
		max_subsampling_layer_testing_schema::max_subsampling_layer_testing_schema()
		{
		}

		max_subsampling_layer_testing_schema::~max_subsampling_layer_testing_schema()
		{
		}

		const boost::uuids::uuid& max_subsampling_layer_testing_schema::get_uuid() const
		{
			return max_subsampling_layer::layer_guid;
		}

		layer_testing_schema_smart_ptr max_subsampling_layer_testing_schema::create_specific() const
		{
			return layer_testing_schema_smart_ptr(new max_subsampling_layer_testing_schema());
		}

		layer_tester_cuda_smart_ptr max_subsampling_layer_testing_schema::create_tester_specific(
			const layer_configuration_specific& input_configuration_specific,
			const layer_configuration_specific& output_configuration_specific) const
		{
			layer_tester_cuda_smart_ptr res;

			switch (output_configuration_specific.dimension_sizes.size())
			{
				case 1: 
					res = layer_tester_cuda_smart_ptr(new max_subsampling_layer_tester_cuda<1>());
					break;
				case 2:
					res = layer_tester_cuda_smart_ptr(new max_subsampling_layer_tester_cuda<2>());
					break;
				case 3:
					res = layer_tester_cuda_smart_ptr(new max_subsampling_layer_tester_cuda<3>());
					break;
				case 4:
					res = layer_tester_cuda_smart_ptr(new max_subsampling_layer_tester_cuda<4>());
					break;
				default:
					throw neural_network_exception((boost::format("No CUDA tester for the max subsampling of %1% dimensions") % output_configuration_specific.dimension_sizes.size()).str());
			}

			return res;
		}
	}
}
