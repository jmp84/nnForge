/*
 *  Copyright 2011-2013 Maxim Milakov
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include "layer_updater_cuda.h"

#include "util_cuda.h"
#include "neural_network_cuda_exception.h"

namespace nnforge
{
	namespace cuda
	{
		layer_updater_cuda::layer_updater_cuda()
		{
		}

		layer_updater_cuda::~layer_updater_cuda()
		{
		}

		void layer_updater_cuda::configure(
			const layer_configuration_specific& input_configuration_specific,
			const layer_configuration_specific& output_configuration_specific,
			const_layer_smart_ptr layer_schema,
			cuda_running_configuration_const_smart_ptr cuda_config,
			bool backprop_required,
			bool different_input)
		{
			this->layer_schema = layer_schema;
			this->input_configuration_specific = input_configuration_specific;
			this->output_configuration_specific = output_configuration_specific;
			this->cuda_config = cuda_config;
			this->backprop_required = backprop_required;
			this->different_input = different_input;

			input_elem_count_per_entry = input_configuration_specific.get_neuron_count();
			output_elem_count_per_entry = output_configuration_specific.get_neuron_count();
			input_elem_count_per_feature_map = input_configuration_specific.get_neuron_count_per_feature_map();
			output_elem_count_per_feature_map = output_configuration_specific.get_neuron_count_per_feature_map();

			updater_configured();
		}

		void layer_updater_cuda::updater_configured()
		{
		}

		std::vector<size_t> layer_updater_cuda::get_sizes_of_additional_buffers_per_entry() const
		{
			return std::vector<size_t>();
		}

		std::vector<unsigned int> layer_updater_cuda::get_linear_addressing_through_texture_per_entry() const
		{
			return std::vector<unsigned int>();
		}

		void layer_updater_cuda::update_buffer_configuration(buffer_cuda_size_configuration& buffer_configuration) const
		{
			std::vector<size_t> per_entry_sizes = get_sizes_of_additional_buffers_per_entry();
			for(std::vector<size_t>::const_iterator it = per_entry_sizes.begin(); it != per_entry_sizes.end(); ++it)
				buffer_configuration.add_per_entry_buffer(*it);

			std::vector<size_t> fixed_sized = get_sizes_of_additional_buffers_fixed();
			for(std::vector<size_t>::const_iterator it = fixed_sized.begin(); it != fixed_sized.end(); ++it)
				buffer_configuration.add_constant_buffer(*it);

			buffer_configuration.add_per_entry_buffer(output_elem_count_per_entry * sizeof(float));

			if (backprop_required && !is_in_place_backprop())
				buffer_configuration.add_per_entry_buffer(input_elem_count_per_entry * sizeof(float));

			std::vector<unsigned int> tex_per_entry = get_linear_addressing_through_texture_per_entry();
			for(std::vector<unsigned int>::const_iterator it = tex_per_entry.begin(); it != tex_per_entry.end(); ++it)
				buffer_configuration.add_per_entry_linear_addressing_through_texture(*it);
		}

		void layer_updater_cuda::update_buffer_configuration(
			buffer_cuda_size_configuration& buffer_configuration,
			unsigned int updater_entry_count) const
		{
			std::vector<size_t> per_entry_sizes = get_sizes_of_additional_buffers_per_entry();
			for(std::vector<size_t>::const_iterator it = per_entry_sizes.begin(); it != per_entry_sizes.end(); ++it)
				buffer_configuration.add_constant_buffer(*it * updater_entry_count);

			std::vector<size_t> fixed_sizes = get_sizes_of_additional_buffers_fixed();
			for(std::vector<size_t>::const_iterator it = fixed_sizes.begin(); it != fixed_sizes.end(); ++it)
				buffer_configuration.add_constant_buffer(*it);

			buffer_configuration.add_constant_buffer(output_elem_count_per_entry * sizeof(float) * updater_entry_count);

			if (backprop_required && !is_in_place_backprop())
				buffer_configuration.add_constant_buffer(input_elem_count_per_entry * sizeof(float) * updater_entry_count);
		}

		layer_updater_cuda::buffer_set layer_updater_cuda::allocate_all_buffers(unsigned int max_entry_count)
		{
			buffer_set res;

			set_max_entry_count(max_entry_count);

			std::vector<size_t> per_entry_sizes = get_sizes_of_additional_buffers_per_entry();
			for(std::vector<size_t>::const_iterator it = per_entry_sizes.begin(); it != per_entry_sizes.end(); ++it)
				res.additional_buffers.push_back(cuda_linear_buffer_device_smart_ptr(new cuda_linear_buffer_device(*it * max_entry_count)));

			std::vector<size_t> fixed_sizes = get_sizes_of_additional_buffers_fixed();
			for(std::vector<size_t>::const_iterator it = fixed_sizes.begin(); it != fixed_sizes.end(); ++it)
				res.additional_buffers.push_back(cuda_linear_buffer_device_smart_ptr(new cuda_linear_buffer_device(*it)));

			fill_additional_buffers(res.additional_buffers);

			{
				size_t sz = output_elem_count_per_entry * sizeof(float) * max_entry_count;
				res.output_neurons_buffer = cuda_linear_buffer_device_smart_ptr(new cuda_linear_buffer_device(sz));
			}

			if (backprop_required && !is_in_place_backprop())
			{
				size_t sz = input_elem_count_per_entry * sizeof(float) * max_entry_count;
				res.input_errors_buffer = cuda_linear_buffer_device_smart_ptr(new cuda_linear_buffer_device(sz));
			}

			res.dynamic_memobjects.resize(get_dynamic_memobject_count());

			return res;
		}

		void layer_updater_cuda::enqueue_update_weights(
			unsigned int offset_input_entry_id,
			hipStream_t stream_id,
			const std::vector<cuda_linear_buffer_device_smart_ptr>& data,
			const std::vector<const_cuda_linear_buffer_device_smart_ptr>& schema_data,
			const std::vector<const_cuda_linear_buffer_device_smart_ptr>& learning_rate,
			cuda_linear_buffer_device_smart_ptr output_errors_buffer,
			const_cuda_linear_buffer_device_smart_ptr input_neurons_buffer,
			const std::vector<cuda_linear_buffer_device_smart_ptr>& additional_buffers,
			std::vector<cuda_memobject_smart_ptr>& dynamic_memobjects,
			unsigned int entry_count,
			float weight_decay)
		{
		}

		void layer_updater_cuda::fill_additional_buffers(const std::vector<cuda_linear_buffer_device_smart_ptr>& additional_buffers) const
		{
		}

		std::vector<size_t> layer_updater_cuda::get_sizes_of_additional_buffers_fixed() const
		{
			return std::vector<size_t>();
		}

		void layer_updater_cuda::set_max_entry_count(unsigned int max_entry_count)
		{
		}

		int layer_updater_cuda::get_dynamic_memobject_count() const
		{
			return 0;
		}

		std::vector<cuda_linear_buffer_device_smart_ptr> layer_updater_cuda::get_data(const std::vector<layer_data_smart_ptr>& host_data_list) const
		{
			std::vector<cuda_linear_buffer_device_smart_ptr> res;

			unsigned int part_count = host_data_list.front()->size();
			for(unsigned int subindex = 0; subindex< part_count; ++subindex)
			{
				unsigned int single_size = get_data_elem_count(subindex, host_data_list.front()->at(subindex).size());
				std::vector<float> pack(single_size * host_data_list.size());

				std::vector<float>::iterator fill_it = pack.begin();
				for(std::vector<layer_data_smart_ptr>::const_iterator sample_it = host_data_list.begin(); sample_it != host_data_list.end(); ++sample_it, fill_it += single_size)
				{
					const std::vector<float>& inp_buf = (*sample_it)->at(subindex);
					fill_data_for_device(subindex, &(*inp_buf.begin()), &(*fill_it), inp_buf.size());
				}

				res.push_back(cuda_linear_buffer_device_smart_ptr(new cuda_linear_buffer_device(
					&(*pack.begin()),
					pack.size() * sizeof(float))));
			}

			return res;
		}

		std::vector<const_cuda_linear_buffer_device_smart_ptr> layer_updater_cuda::get_learning_rate(const std::vector<const_layer_data_smart_ptr>& host_learning_rate_list) const
		{
			std::vector<const_cuda_linear_buffer_device_smart_ptr> res;

			unsigned int part_count = host_learning_rate_list.front()->size();
			for(unsigned int subindex = 0; subindex< part_count; ++subindex)
			{
				unsigned int single_size = get_data_elem_count(subindex, host_learning_rate_list.front()->at(subindex).size());
				std::vector<float> pack(single_size * host_learning_rate_list.size());

				std::vector<float>::iterator fill_it = pack.begin();
				for(std::vector<const_layer_data_smart_ptr>::const_iterator sample_it = host_learning_rate_list.begin(); sample_it != host_learning_rate_list.end(); ++sample_it, fill_it += single_size)
				{
					const std::vector<float>& inp_buf = (*sample_it)->at(subindex);
					fill_data_for_device(subindex, &(*inp_buf.begin()), &(*fill_it), inp_buf.size());
				}

				res.push_back(const_cuda_linear_buffer_device_smart_ptr(new cuda_linear_buffer_device(
					&(*pack.begin()),
					pack.size() * sizeof(float))));
			}

			return res;
		}

		void layer_updater_cuda::get_data_from_device(const std::vector<cuda_linear_buffer_device_smart_ptr>& device_data, std::vector<layer_data_smart_ptr>& host_data) const
		{
			unsigned int part_count = host_data.front()->size();
			std::vector<cuda_linear_buffer_device_smart_ptr>::const_iterator src_it = device_data.begin();
			for(unsigned int subindex = 0; subindex< part_count; ++subindex, ++src_it)
			{
				unsigned int single_size = get_data_elem_count(subindex, host_data.front()->at(subindex).size());
				cuda_linear_buffer_device_smart_ptr src = *src_it;
				std::vector<float> pack(src->get_size() / sizeof(float));
				cuda_safe_call(hipMemcpy(&(*pack.begin()), *src, pack.size() * sizeof(float), hipMemcpyDeviceToHost));

				std::vector<float>::const_iterator src_buf_it = pack.begin();
				for(std::vector<layer_data_smart_ptr>::const_iterator sample_it = host_data.begin(); sample_it != host_data.end(); ++sample_it, src_buf_it += single_size)
				{
					std::vector<float>& dst_buf = (*sample_it)->at(subindex);
					fill_data_for_host(subindex, &(*src_buf_it), &(*dst_buf.begin()), single_size);
				}
			}
		}

		unsigned int layer_updater_cuda::get_data_elem_count(unsigned int part_id, unsigned int source_elem_count) const
		{
			return source_elem_count;
		}

		void layer_updater_cuda::fill_data_for_device(
			unsigned int part_id,
			const float * src,
			float * dst,
			unsigned int count) const
		{
			std::copy(src, src + count, dst);
		}

		void layer_updater_cuda::fill_data_for_host(
			unsigned int part_id,
			const float * src,
			float * dst,
			unsigned int count) const
		{
			std::copy(src, src + count, dst);
		}

		std::vector<unsigned int> layer_updater_cuda::get_incoming_weight_count_per_output_neuron_list() const
		{
			return std::vector<unsigned int>();
		}
	}
}
