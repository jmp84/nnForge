#include "hip/hip_runtime.h"
/*
 *  Copyright 2011-2013 Maxim Milakov
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include "softmax_layer_updater_cuda.h"

#include <hip/hip_runtime.h>

#include "../neural_network_exception.h"
#include "neural_network_cuda_exception.h"

#include "util_cuda.h"

/**
 * computes max using the reduce algorithm
 */
__global__ void max_kernel(
		const float * __restrict input,
		float * __restrict output,
		unsigned int size) {

	unsigned int tid = threadIdx.x;
	unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;

	// global to shared memory
	// if out of bounds, use identity elt (here it's a max over exps,
	// so zero is fine)
	extern __shared__ float sdata[];
	sdata[tid] = id < size ? input[id] : 0.0F;
	__syncthreads();

	// reduction algorithm
	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
		if (tid < s) {
			sdata[tid] = max(sdata[tid], sdata[tid + s]);
		}
		__syncthreads();
	}

	// store the block max into the output
	if (tid == 0) {
		output[blockIdx.x] = sdata[0];
	}
}

/**
 * computes exp(x_i - m) over a range
 * m is max(x_i)
 */
__global__ void exponential_minus_max_kernel(
		const float * __restrict input,
		float * __restrict output,
		const float* max_exp,
		unsigned int size) {

	unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;

	// check out of bounds
	if (id >= size) {
		return;
	}
	output[id] = expf(input[id] - *max_exp);
}

/**
 * computes sum using the reduce algorithm
 */
__global__ void sum_kernel(
		const float * __restrict input,
		float * __restrict output,
		unsigned int size) {

	unsigned int tid = threadIdx.x;
	unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;

	// global to shared memory
	// if out of bounds, use identity elt for sum, i.e. 0
	extern __shared__ float sdata[];
	sdata[tid] = id < size ? input[id] : 0.0F;
	__syncthreads();

	// reduction algorithm
	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
		if (tid < s) {
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}

	// store the block sum into the output
	if (tid == 0) {
		output[blockIdx.x] = sdata[0];
	}
}

/**
 * divides all elements in a range by a normalizing constant
 */
__global__ void normalize_kernel(
		float * __restrict output,
		float* sum,
		unsigned int size) {

	unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;

	// check out of bounds
	if (id >= size) {
		return;
	}

	output[id] /= (*sum);
}

__global__ void dot_product_kernel(
		const float * __restrict input1,
		const float * __restrict input2,
		float* __restrict output,
		unsigned int size) {

	unsigned int tid = threadIdx.x;
	unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;

	// global to shared memory, elt by elt multiplication
	// if out of bounds, use identity elt
	extern __shared__ float sdata[];
	sdata[tid] = (id < size) ? (input1[id] * input2[id]) : 0.0F;
	__syncthreads();

	// reduction algorithm
	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
		if (tid < s) {
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}

	// store the block dot product into the output
	if (tid == 0) {
		output[blockIdx.x] = sdata[0];
	}
}

__global__ void elementwise_mult_minus_sum_kernel(
		const float* __restrict input,
		float* __restrict output,
		const float* sum,
		unsigned int size) {

	unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;

	// check out of bounds
	if (id >= size) {
		return;
	}

	output[id] = input[id] * (output[id] - *sum);
}

__global__ void softmax_deriviative_upd_kernel(
	float * __restrict errors,
	const float * __restrict output_neurons,
	int feature_map_count,
	int neuron_count_per_feature_map,
	int entry_count)
{
	int neuron_id = blockIdx.x * blockDim.x + threadIdx.x;
	int entry_id = blockIdx.y * blockDim.y + threadIdx.y;
	if ((neuron_id < neuron_count_per_feature_map) && (entry_id < entry_count))
	{
		int initial_offset = entry_id * feature_map_count * neuron_count_per_feature_map + neuron_id;
		float sum = 0.0F;
		const float * current_output_neurons = output_neurons + initial_offset;
		const float * current_output_errors = errors + initial_offset;
		for(int i = 0; i < feature_map_count; ++i)
		{
			sum += __load_nc(current_output_neurons) * __load_nc(current_output_errors);
			current_output_neurons += neuron_count_per_feature_map;
			current_output_errors += neuron_count_per_feature_map;
		}

		current_output_neurons = output_neurons + initial_offset;
		float * current_errors = errors + initial_offset;
		for(int i = 0; i < feature_map_count; ++i)
		{
			*current_errors = __load_nc(current_output_neurons) * (__load_nc(current_errors) - sum);
			current_output_neurons += neuron_count_per_feature_map;
			current_errors += neuron_count_per_feature_map;
		}
	}
}

namespace nnforge
{
	namespace cuda
	{
		softmax_layer_updater_cuda::softmax_layer_updater_cuda()
		{
		}

		softmax_layer_updater_cuda::~softmax_layer_updater_cuda()
		{
		}

		// from http://graphics.stanford.edu/~seander/bithacks.html
		unsigned int next_power_of_two(unsigned int const v) {
			if (sizeof(unsigned int) != 4) {
				throw neural_network_exception(
						"Size of unsigned int is not 4, needed for bit hacks");
			}
			if (v > 1) {
				float f = (float)v;
				unsigned int const t = 1U << ((*(unsigned int *)&f >> 23) - 0x7f);
				return t << (t < v);
			} else {
				return 1;
			}
			// should not read here
			return 0;
		}

		/**
		 * computes softmax
		 * softmax(x_1, ..., x_n) = (x_1/sum(x_i), ..., x_n/sum(x_i)
		 * to avoid underflow/overflow, we use the log sum exp trick
		 * (http://math.stackexchange.com/questions/648514/preventing-underflow-log-sum-exp-trick)
		 * softmax is parallelized for speed (in nnjm paper, layer size is 32000)
		 */
		void softmax_layer_updater_cuda::enqueue_test(
			unsigned int offset_input_entry_id,
			hipStream_t stream_id,
			const std::vector<const_cuda_linear_buffer_device_smart_ptr>& schema_data,
			const std::vector<cuda_linear_buffer_device_smart_ptr>& data,
			const_cuda_linear_buffer_device_smart_ptr input_neurons_buffer,
			cuda_linear_buffer_device_smart_ptr output_neurons_buffer,
			const std::vector<cuda_linear_buffer_device_smart_ptr>& additional_buffers,
			std::vector<cuda_memobject_smart_ptr>& dynamic_memobjects,
			unsigned int entry_count)
		{
			// max number of thread on K20 gpu
			const unsigned int NUM_THREADS = 1024;
			unsigned int numThreads(NUM_THREADS);
			unsigned int layer_size =
					input_elem_count_per_feature_map *
					input_configuration_specific.feature_map_count;
			unsigned int remainder = layer_size % NUM_THREADS;
			unsigned int quotient = layer_size / NUM_THREADS;
			unsigned int numBlocks = remainder == 0 ? quotient : quotient + 1;
			unsigned int size_shared_memory = sizeof(float) * NUM_THREADS;
			unsigned int size_global_memory = sizeof(float) * layer_size;
			unsigned int numBlocksPower2 = next_power_of_two(numBlocks);
			unsigned int size_intermediate_shared_memory =
					sizeof(float) * numBlocksPower2;
			if (offset_input_entry_id > 0)
				throw neural_network_exception("softmax_layer_updater_cuda is not able to run using offset");


			float* max_intermediate;
			float* max_global;
			float* exp_minus_max;
			float* sum_exps_intermediate;
			float* sum_exps;
			cuda_safe_call(hipMalloc(
					(void **) &max_intermediate,
					sizeof(float) * numBlocks));
			cuda_safe_call(hipMalloc((void **) &max_global, sizeof(float)));
			cuda_safe_call(hipMalloc((void **) &exp_minus_max, size_global_memory));
			cuda_safe_call(hipMalloc(
					(void **) &sum_exps_intermediate,
					sizeof(float) * numBlocks));
			cuda_safe_call(hipMalloc((void **) &sum_exps, sizeof(float)));

			// compute max
			// only 2 calls, assume size <= than 1024 x 1024
			// first: max per block
			max_kernel<<<numBlocks, numThreads, size_shared_memory>>>(
					*input_neurons_buffer,
					max_intermediate,
					layer_size);
			// second: global max
			max_kernel<<<1, numBlocksPower2, size_intermediate_shared_memory>>>(
					max_intermediate,
					max_global,
					numBlocks);

			// compute exp subtracting the max
			exponential_minus_max_kernel<<<numBlocks, numThreads>>>(
					*input_neurons_buffer,
					*output_neurons_buffer,
					max_global,
					layer_size);

			// compute the sum
			// only 2 calls, assume size <= 1024 x 1024
			// first: sum per block
			sum_kernel<<<numBlocks, numThreads, size_shared_memory>>>(
					*output_neurons_buffer,
					sum_exps_intermediate,
					layer_size);
			// second: global sum
			sum_kernel<<<1, numBlocksPower2, size_intermediate_shared_memory>>>(
					sum_exps_intermediate,
					sum_exps,
					numBlocks);

			// finally normalize
			normalize_kernel<<<numBlocks, numThreads>>> (
					*output_neurons_buffer,
					sum_exps,
					layer_size);

			// clean up
			cuda_safe_call(hipFree(max_intermediate));
			cuda_safe_call(hipFree(max_global));
			cuda_safe_call(hipFree(exp_minus_max));
			cuda_safe_call(hipFree(sum_exps_intermediate));
			cuda_safe_call(hipFree(sum_exps));
		}

		void softmax_layer_updater_cuda::enqueue_backprop(
			hipStream_t stream_id,
			const std::vector<const_cuda_linear_buffer_device_smart_ptr>& schema_data,
			const std::vector<cuda_linear_buffer_device_smart_ptr>& data,
			const_cuda_linear_buffer_device_smart_ptr output_neurons_buffer,
			const_cuda_linear_buffer_device_smart_ptr input_neurons_buffer,
			cuda_linear_buffer_device_smart_ptr output_errors_buffer,
			cuda_linear_buffer_device_smart_ptr input_errors_buffer,
			const std::vector<cuda_linear_buffer_device_smart_ptr>& additional_buffers,
			std::vector<cuda_memobject_smart_ptr>& dynamic_memobjects,
			unsigned int entry_count)
		{
			std::pair<dim3, dim3> kernel_dims = cuda_util::get_grid_and_threadblock_sizes_sequential_access(
				*cuda_config,
				input_elem_count_per_feature_map,
				entry_count,
				1);

			// max number of thread on K20 gpu
			const unsigned int NUM_THREADS = 1024;
			unsigned int numThreads(NUM_THREADS);
			unsigned int layer_size =
					input_elem_count_per_feature_map *
					input_configuration_specific.feature_map_count;
			unsigned int remainder = layer_size % NUM_THREADS;
			unsigned int quotient = layer_size / NUM_THREADS;
			unsigned int numBlocks = remainder == 0 ? quotient : quotient + 1;
			unsigned int size_shared_memory = sizeof(float) * NUM_THREADS;
			unsigned int size_global_memory = sizeof(float) * layer_size;
			unsigned int numBlocksPower2 = next_power_of_two(numBlocks);
			unsigned int size_intermediate_shared_memory =
					sizeof(float) * numBlocksPower2;

			float* intermediate_dot_product;
			float* final_dot_product;
			hipMalloc((void**) &intermediate_dot_product, sizeof(float) * numBlocks);
			hipMalloc((void**) &final_dot_product, sizeof(float));

			// compute dot product
			dot_product_kernel<<<numBlocks, numThreads, size_shared_memory>>>(
					*output_errors_buffer,
					*output_neurons_buffer,
					intermediate_dot_product,
					layer_size);
			sum_kernel<<<1, numBlocksPower2, size_intermediate_shared_memory>>>(
					intermediate_dot_product,
					final_dot_product,
					numBlocks);

			// compute elementwise multiplication, dot product
			// subtracted from second operand
			elementwise_mult_minus_sum_kernel<<<numBlocks, numThreads>>>(
					*output_neurons_buffer,
					*output_errors_buffer,
					final_dot_product,
					layer_size);

			// clean up
			cuda_safe_call(hipFree(intermediate_dot_product));
			cuda_safe_call(hipFree(final_dot_product));
		}

		bool softmax_layer_updater_cuda::is_in_place_backprop() const
		{
			return true;
		}
	}
}
