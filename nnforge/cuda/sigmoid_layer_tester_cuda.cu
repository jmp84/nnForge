#include "hip/hip_runtime.h"
/*
 *  Copyright 2011-2013 Maxim Milakov
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include "sigmoid_layer_tester_cuda.h"

#include <hip/hip_runtime.h>

#include "util_cuda.h"

#include "../nn_types.h"

static __forceinline__ __device__ float sigmoid(float x)
{
	return __fdividef(1.0F, 1.0F + __expf(-x));
}

__global__ void sigmoid_kernel(
	float4 * __restrict input,
	int elem_count)
{
	int elem_id = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;
	if (elem_id < elem_count)
	{
		float4 val = input[elem_id];
		val.x = sigmoid(val.x);
		val.y = sigmoid(val.y);
		val.z = sigmoid(val.z);
		val.w = sigmoid(val.w);
		input[elem_id] = val;
	}
}

namespace nnforge
{
	namespace cuda
	{
		sigmoid_layer_tester_cuda::sigmoid_layer_tester_cuda()
		{
		}

		sigmoid_layer_tester_cuda::~sigmoid_layer_tester_cuda()
		{
		}

		void sigmoid_layer_tester_cuda::enqueue_test(
			hipStream_t stream_id,
			const std::vector<const_cuda_linear_buffer_device_smart_ptr>& schema_data,
			const std::vector<const_cuda_linear_buffer_device_smart_ptr>& data,
			cuda_linear_buffer_device_smart_ptr input_buffer,
			const std::vector<cuda_linear_buffer_device_smart_ptr>& additional_buffers,
			unsigned int entry_count)
		{
			int elem_count = (input_elem_count_per_entry * entry_count + 3) / 4;
			std::pair<dim3, dim3> kernel_dims = cuda_util::get_grid_and_threadblock_sizes_sequential_access(
				*cuda_config,
				elem_count);
			sigmoid_kernel<<<kernel_dims.first, kernel_dims.second, 0, stream_id>>>(
				*input_buffer,
				elem_count);
		}
	}
}
