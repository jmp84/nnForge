#include "hip/hip_runtime.h"
/*
 *  Copyright 2011-2013 Maxim Milakov
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include "absolute_layer_updater_cuda.h"

#include <hip/hip_runtime.h>

#include "../neural_network_exception.h"
#include "util_cuda.h"


__global__ void absolute_upd_kernel(
	const float4 * __restrict input,
	float4 * __restrict output,
	int elem_count)
{
	int elem_id = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;
	if (elem_id < elem_count)
	{
		float4 val = input[elem_id];
		val.x = fabsf(val.x);
		val.y = fabsf(val.y);
		val.z = fabsf(val.z);
		val.w = fabsf(val.w);
		output[elem_id] = val;
	}
}

__global__ void absolute_deriviative_upd_kernel(
	float4 * __restrict errors,
	const float4 * __restrict input_neurons,
	int elem_count)
{
	int elem_id = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;
	if (elem_id < elem_count)
	{
		float4 inp = input_neurons[elem_id];
		float4 current_error = errors[elem_id];
		if (inp.x < 0.0F)
			current_error.x = -current_error.x;
		if (inp.y < 0.0F)
			current_error.y = -current_error.y;
		if (inp.z < 0.0F)
			current_error.z = -current_error.z;
		if (inp.w < 0.0F)
			current_error.w = -current_error.w;
		errors[elem_id] = current_error;
	}
}

namespace nnforge
{
	namespace cuda
	{
		absolute_layer_updater_cuda::absolute_layer_updater_cuda()
		{
		}

		absolute_layer_updater_cuda::~absolute_layer_updater_cuda()
		{
		}

		void absolute_layer_updater_cuda::enqueue_test(
			unsigned int offset_input_entry_id,
			hipStream_t stream_id,
			const std::vector<const_cuda_linear_buffer_device_smart_ptr>& schema_data,
			const std::vector<cuda_linear_buffer_device_smart_ptr>& data,
			const_cuda_linear_buffer_device_smart_ptr input_neurons_buffer,
			cuda_linear_buffer_device_smart_ptr output_neurons_buffer,
			const std::vector<cuda_linear_buffer_device_smart_ptr>& additional_buffers,
			std::vector<cuda_memobject_smart_ptr>& dynamic_memobjects,
			unsigned int entry_count)
		{
			if (offset_input_entry_id > 0)
				throw neural_network_exception("absolute_layer_updater_cuda is not able to run using offset");

			int elem_count = (input_elem_count_per_entry * entry_count + 3) / 4;
			std::pair<dim3, dim3> kernel_dims = cuda_util::get_grid_and_threadblock_sizes_sequential_access(
				*cuda_config,
				elem_count);
			absolute_upd_kernel<<<kernel_dims.first, kernel_dims.second, 0, stream_id>>>(
				*input_neurons_buffer,
				*output_neurons_buffer,
				elem_count);
		}

		void absolute_layer_updater_cuda::enqueue_backprop(
			hipStream_t stream_id,
			const std::vector<const_cuda_linear_buffer_device_smart_ptr>& schema_data,
			const std::vector<cuda_linear_buffer_device_smart_ptr>& data,
			const_cuda_linear_buffer_device_smart_ptr output_neurons_buffer,
			const_cuda_linear_buffer_device_smart_ptr input_neurons_buffer,
			cuda_linear_buffer_device_smart_ptr output_errors_buffer,
			cuda_linear_buffer_device_smart_ptr input_errors_buffer,
			const std::vector<cuda_linear_buffer_device_smart_ptr>& additional_buffers,
			std::vector<cuda_memobject_smart_ptr>& dynamic_memobjects,
			unsigned int entry_count)
		{
			int elem_count = (input_elem_count_per_entry * entry_count + 3) / 4;
			std::pair<dim3, dim3> kernel_dims = cuda_util::get_grid_and_threadblock_sizes_sequential_access(
				*cuda_config,
				elem_count);
			absolute_deriviative_upd_kernel<<<kernel_dims.first, kernel_dims.second, 0, stream_id>>>(
				*output_errors_buffer,
				*input_neurons_buffer,
				elem_count);
		}

		bool absolute_layer_updater_cuda::is_in_place_backprop() const
		{
			return true;
		}
	}
}
